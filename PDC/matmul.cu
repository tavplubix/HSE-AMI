#include <chrono>
#include <iostream>

#include <hip/hip_runtime.h>

using namespace std::chrono;
using namespace std;

void gen_matrix(double *matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = double(rand()) / RAND_MAX;
    }
}

// === DO NOT REMOVE THIS LINE ===

// Insert your matrix multiplication kernel below

#define IDX(row, col, len) ((row)*(len)+(col))

// A - KxM, B - MxN, C - KxN
__host__ void matmul_cpu(double *A, double *B, double *C, int K, int M, int N) {
    for (int i = 0; i < K; ++i) {
        for (int j = 0; j < N; ++j) {
            int c_idx = IDX(i, j, N);
            C[c_idx] = 0.0;
            for (int k = 0; k < M; ++k) {
                int a_idx = IDX(i, k, M);
                int b_idx = IDX(k, j, N);
                C[c_idx] += A[a_idx] * B[b_idx];
            }
        }
    }
}

// === DO NOT REMOVE THIS LINE ===

int main(int argc, char* argv[]) {
    if (argc != 4) {
        cerr << "Usage: " << argv[0] << " K M N" << endl;
        return 1;
    }
    unsigned K = atoi(argv[1]);
    unsigned M = atoi(argv[2]);
    unsigned N = atoi(argv[3]);
    srand(K + M + N);

    cout << "A: " << K << "x" << M << endl;
    cout << "B: " << M << "x" << N << endl;
    cout << "C: " << K << "x" << N << endl;
    cout << endl;

    double *A = new double[K * M];
    double *B = new double[M * N];

    gen_matrix(A, K, M);
    gen_matrix(B, M, N);

    double *C = new double[K * N];
    double *cpuC = new double[K * N];

    high_resolution_clock::time_point total_start = high_resolution_clock::now();

    // === DO NOT REMOVE THIS LINE ===

    // Insert your code that runs C=A*B on GPU below

    matmul_cpu(A, B, cpuC, K, M, N);

    // === DO NOT REMOVE THIS LINE ===

    high_resolution_clock::time_point total_end = high_resolution_clock::now();
    double total_time = duration_cast<duration<double>>(total_end - total_start).count();

    cout << "Total (kernel+copy) time: " << total_time << endl;

    double maxdiff = 0;
    for (unsigned i = 0; i < K*N; ++i)
            maxdiff = std::max(std::abs(cpuC[i] - C[i]), maxdiff);

    cout << "Max diff " << maxdiff << endl;

    delete A;
    delete B;
    delete C;
}
